#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>

#include "flamegpu/flame_api.h"

/**
 * Simple Model demosntrating concurrency within a FLAMEGPU2 model, using mesasge lists and DAG based specificaiton.
 *  
 */

const int CONCURRENCY_DEGREE = 4;

// const int WARMUP_REPETITIONS = 1;
// const int TIMING_REPETITIONS = 3;
const int WARMUP_REPETITIONS = 0;
const int TIMING_REPETITIONS = 1;

// const float SPEEDUP_THRESHOLD = 1.5;


/** 
 * Utility function to time N repetitions of a simulation, returning the mean (but skipping the first)
 */
float meanSimulationTime(const int WARMUP_REPETITIONS, const int REPETITIONS, CUDASimulation &s, std::vector<AgentVector *> const &populations) {
    float total_time = 0.f;
    for (int r = 0; r < REPETITIONS + WARMUP_REPETITIONS; r++) {
        // re-set each population
        for (AgentVector* pop : populations) {
            s.setPopulationData(*pop);
        }
        // Run and time the simulation
        s.simulate();
        // Store the time if not the 0th rep of the model.
        if (r >= WARMUP_REPETITIONS) {
            total_time += s.getElapsedTimeSimulation();
        }
    }
    return total_time / REPETITIONS;
}

/** 
 * Utility function checking for a speedup after running a sim with and without concurrency.
 */
float concurrentLayerSpeedup(const int WARMUP_REPETITIONS, const int REPETITIONS, CUDASimulation &s, std::vector<AgentVector*> const &populations) {
    // Set a single step.
    s.SimulationConfig().steps = 1;

    // Set the flag saying don't use concurrency.
    s.CUDAConfig().inLayerConcurrency = false;
    s.applyConfig();
    // EXPECT_EQ(s.CUDAConfig().inLayerConcurrency, false);

    // Time the simulation multiple times to get an average
    float mean_sequential_time = meanSimulationTime(WARMUP_REPETITIONS, REPETITIONS, s, populations);

    // set the flag saying to use streams for agnet function concurrency.
    s.CUDAConfig().inLayerConcurrency = true;
    s.applyConfig();
    // EXPECT_EQ(s.CUDAConfig().inLayerConcurrency, true);

    float mean_concurrent_time = meanSimulationTime(WARMUP_REPETITIONS, REPETITIONS, s, populations);

    printf("mean_sequential_time %f ms\n", mean_sequential_time);
    printf("mean_concurrent_time %f ms\n", mean_concurrent_time);

    // Calculate a speedup value.
    float speedup = mean_sequential_time / mean_concurrent_time;
    return speedup;
}

/**
 * Agent function which inputs from a Spatial3D message list + some slow work.
 * Agents then birth a new agent
 * And then they all die (for a stable population.)
 * This is unrealistic, but demonstrates the problem.
 */
FLAMEGPU_AGENT_FUNCTION(outputBirthDeath, MsgNone, MsgSpatial3D) {
    // Repeatedly do some pointless maths on the value in register
    // const int INTERNAL_REPETITIONS = 65536;
    const int INTERNAL_REPETITIONS = 4096; // Need to make the kernel long enough to ensure that concurrency is actually observed.
    for (int i = 0; i < INTERNAL_REPETITIONS; i++) {
        // Read and write all the way to global mem each time to make this intentionally slow
        float v = FLAMEGPU->getVariable<float>("v");
        FLAMEGPU->setVariable("v", v + v);
    }
    FLAMEGPU->message_out.setVariable("v", FLAMEGPU->getVariable<float>("v"));
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"),
        FLAMEGPU->getVariable<float>("z")
    );

    // Birth
    FLAMEGPU->agent_out.setVariable<float>("v", FLAMEGPU->getVariable<float>("v"));
    FLAMEGPU->agent_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
    FLAMEGPU->agent_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
    FLAMEGPU->agent_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));

    // Death
    return DEAD;
}

/**
 * Agent function which inputs from a Spatial3D message list + some slow work.
 */
FLAMEGPU_AGENT_FUNCTION(intput, MsgSpatial3D, MsgNone) {
    // Repeatedly do some pointless maths on the value in register
    // const int INTERNAL_REPETITIONS = 65536;
    const int INTERNAL_REPETITIONS = 4096; // Need to make the kernel long enough to ensure that concurrency is actually observed.
    for (int i = 0; i < INTERNAL_REPETITIONS; i++) {
        // Read and write all the way to global mem each time to make this intentionally slow
        float v = FLAMEGPU->getVariable<float>("v");
        FLAMEGPU->setVariable("v", v + v);
    }
    float vSum = 0.f;
    float agent_x = FLAMEGPU->getVariable<float>("x");
    float agent_y = FLAMEGPU->getVariable<float>("y");
    float agent_z = FLAMEGPU->getVariable<float>("z");
    for (const auto &message : FLAMEGPU->message_in(agent_x, agent_y, agent_z)) {
        vSum += message.getVariable<float>("v");
    }
    FLAMEGPU->setVariable("v", vSum);
    return ALIVE;
}


unsigned int fullUtilisationThreadCount(const int deviceIdx) {
    // Find the number of threads to max out the device if 100% utilisation is achieved.#
    hipError_t status;
    int multiprocessors = 0;
    int maxThreadsPerSM = 0;
    status = hipDeviceGetAttribute(&multiprocessors, hipDeviceAttributeMultiprocessorCount, deviceIdx);
    if(hipSuccess != status) {
        fprintf(stdout, "Erorr getting hipDeviceAttributeMultiprocessorCount. %s:%d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    status = hipDeviceGetAttribute(&maxThreadsPerSM, hipDeviceAttributeMaxThreadsPerMultiProcessor, deviceIdx);
    if(hipSuccess != status) {
        fprintf(stdout, "Erorr getting hipDeviceAttributeMaxThreadsPerMultiProcessor. %s:%d\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    unsigned int threads = multiprocessors * maxThreadsPerSM;
    return threads;
}


int main(int argc, const char ** argv) {

    const unsigned int totalThreads = fullUtilisationThreadCount(0);
    printf("Total Threads required: %u\n", totalThreads);

    // Each pop size is an equal fraction.
    const unsigned int POPULATION_SIZES = totalThreads / CONCURRENCY_DEGREE;
    printf("CONCURRENCY_DEGREE: %u\n", CONCURRENCY_DEGREE);
    printf("POPULATION_SIZES: %u\n", POPULATION_SIZES);

    const float MESSAGE_BOUNDS_MIN = 0.f;
    const float MESSAGE_BOUNDS_MAX = 9.f;
    const float MESSAGE_BOUNDS_RADIUS = 1.f;

    // Define a model with multiple agent types
    ModelDescription m("ConcurrentSpatial3DBirthDeath");

    // Create two layers.
    LayerDescription &layer0  = m.newLayer();
    LayerDescription &layer1  = m.newLayer();

    std::vector<AgentVector*> populations = std::vector<AgentVector*>();

    // Add a few agent types, each with a single agent function.
    for (int i = 0; i < CONCURRENCY_DEGREE; i++) {
        // Generate the agent type
        std::string agent_name("agent_" + std::to_string(i));
        std::string agent_function_out(agent_name + "_outputBirthDeath");
        std::string agent_function_in(agent_name + "_intput");
        std::string message_name(agent_name + "_messages");
        AgentDescription &a = m.newAgent(agent_name);
        a.newVariable<float>("v");
        a.newVariable<float>("x");
        a.newVariable<float>("y");
        a.newVariable<float>("z");

        MsgSpatial3D::Description &msg = m.newMessage<MsgSpatial3D>(message_name);
        msg.newVariable<float>("v");
        msg.setMin(MESSAGE_BOUNDS_MIN, MESSAGE_BOUNDS_MIN, MESSAGE_BOUNDS_MIN);
        msg.setMax(MESSAGE_BOUNDS_MAX, MESSAGE_BOUNDS_MAX, MESSAGE_BOUNDS_MAX);
        msg.setRadius(MESSAGE_BOUNDS_RADIUS);

        auto &f_out = a.newFunction(agent_function_out, outputBirthDeath);
        f_out.setMessageOutput(msg);
        f_out.setAgentOutput(a);
        f_out.setAllowAgentDeath(true);


        layer0.addAgentFunction(f_out);

        auto &f_in = a.newFunction(agent_function_in, intput);
        f_in.setMessageInput(msg);

        layer1.addAgentFunction(f_in);

        // Generate an iniital population.
        AgentVector* a_pop = new AgentVector(a, POPULATION_SIZES);
        // unsigned long int seed = s.getSimulationConfig().random_seed;
        unsigned long int seed = 12; // @todo - fixed seed for now. CLI not yet parsed here  
        std::default_random_engine rng(seed);
        std::uniform_real_distribution<float> dist(0.0f, 11.0f);
        for (unsigned int j = 0; j < POPULATION_SIZES; ++j) {
            auto agent = a_pop->at(j);
            agent.setVariable<float>("v", static_cast<float>(j));
            agent.setVariable<float>("x", dist(rng));
            agent.setVariable<float>("y", dist(rng));
            agent.setVariable<float>("z", dist(rng));
        }
        populations.push_back(a_pop);
    }

    // Convert the model to a simulation
    CUDASimulation s(m);

    // Run the simulation many times, with and without concurrency to get an accurate speedup
    float speedup = concurrentLayerSpeedup(WARMUP_REPETITIONS, TIMING_REPETITIONS, s, populations);
    // Assert that a speedup was achieved.
    printf("speedup %f\n", speedup);


    return EXIT_SUCCESS;
}
